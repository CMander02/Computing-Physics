
#include "hip/hip_runtime.h"

#include<cmath>
#include<iostream>
#include <stdio.h>
#include<iomanip>
#include<fstream>

using namespace std;

#define precision 1E-8
#define PI 3.141592653589793238462643383
#define divide (8192)
#define saving 8192

#define lambdamax 1.5
#define dlambda (double(lambdamax)/divide)

const int times = 1000000;

__global__ void kernel(double *dev_arr)//��ÿһ���߳̽��м���
{
	int offset = blockDim.x*blockIdx.x + threadIdx.x;//λ���ƶ�
	double thread_lambda = offset * dlambda;

	int start = offset * saving;

	for (int i = 0; i < times; i++)
	{
		dev_arr[start + (i) % saving] = thread_lambda * sin(PI * dev_arr[start + (i - 1) % saving]);
	}

}

double arr[divide*saving];//ÿһ���̻߳���һ����СΪ1024������buffer

int main()
{	

	ofstream out("C:\\Users\\10069\\Desktop\\Sinx.txt");
	double *dev_arr;

	hipMalloc((void**)&dev_arr, sizeof(double)*divide*saving);

	for (int i = 0; i < divide*saving; i++)
	{
		arr[i] = rand()-(RAND_MAX/2);
	}
		
	hipMemcpy(dev_arr,arr, sizeof(double)*divide*saving, hipMemcpyHostToDevice);
	

	int blocksize = 512;

	kernel<<<divide / blocksize, blocksize >>>(dev_arr);

	hipMemcpy(arr, dev_arr, sizeof(double)*divide*saving, hipMemcpyDeviceToHost);

	double result = 0;
	out << setprecision(12);
	for (int i = 0; i < divide*saving; i++)
	{
		out << arr[i] << ' ';
	}

	hipFree(dev_arr);
	out.close();

	system("pause");
}

